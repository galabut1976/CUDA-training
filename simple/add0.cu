//: nvcc add0.cu -o add0


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void cuda_add(int a, int b, int *c)
{
    *c = a + b;
}

int main(int argc, char **argv)
{
    int c;
    int *dev_c;
    hipMalloc((void**)&dev_c, sizeof(int));
    cuda_add<<<1,1>>>(2, 7, dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("Almighty CUDA's answer: 2 + 2 = %d.\n", c);
    hipFree(dev_c);
    return EXIT_SUCCESS;
}

